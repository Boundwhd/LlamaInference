#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int id = -1;
    hipError_t state = hipGetDevice(&id);

    if (state == hipSuccess) {
        std::cout << "Current GPU device ID: " << id << std::endl;
    } else {
        std::cerr << "Failed to get device ID: " << hipGetErrorString(state) << std::endl;
    }

    return 0;
}